#include "hip/hip_runtime.h"
/* 
 * File:   increment_gpu.cpp
 * Author: peter 
 *
 * Created on March 26, 2013, 2:58 PM
 */

#include <stdlib.h>
#include <stdio.h>
//#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand_kernel.h>

#include <hip/hip_runtime.h>
// includes, project
//#include <hip/hip_runtime_api.h>
//#include <helper_functions.h> // helper utility functions 

#include "common.h"
#include <algorithm>
#include <vector>
/*
 * 
 */

static __global__ void increment_gpu(float* a, float b, int N){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if( idx < N){
        a[ idx] = a[ idx] + b;
    }
}

void func(){
    printf("jkl");
}

void do_cuda_increment_gpu(DataPlan* plan){ //plan will be stored in device global memory
    increment_gpu<<< plan->dataCount/blockSize, blockSize>>> ( plan->a, plan->increment, plan->dataCount);
    std::vector<int> v(100,0);
    std::sort( v.begin(), v.end());
} 
